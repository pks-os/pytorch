#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/nvrtc_stub/ATenNVRTC.h>
#include <c10/cuda/CUDAGuard.h>

#if !defined(USE_ROCM) && !defined(_WIN32) && defined(CUDA_VERSION) && \
    CUDA_VERSION >= 12000
#define BUILD_ASYNC_MM_KERNEL
#endif

#if defined(BUILD_ASYNC_MM_KERNEL)

// We are going to override the cuTensorMapEncodeTiled driver api with our lazy
// loader
static hipError_t CUDAAPI nvrtc_cuTensorMapEncodeTiled(
    CUtensorMap* tensorMap,
    CUtensorMapDataType tensorDataType,
    cuuint32_t tensorRank,
    void* globalAddress,
    const cuuint64_t* globalDim,
    const cuuint64_t* globalStrides,
    const cuuint32_t* boxDim,
    const cuuint32_t* elementStrides,
    CUtensorMapInterleave interleave,
    CUtensorMapSwizzle swizzle,
    CUtensorMapL2promotion l2Promotion,
    CUtensorMapFloatOOBfill oobFill) {
  return at::globalContext().getNVRTC().cuTensorMapEncodeTiled(
      tensorMap,
      tensorDataType,
      tensorRank,
      globalAddress,
      globalDim,
      globalStrides,
      boxDim,
      elementStrides,
      interleave,
      swizzle,
      l2Promotion,
      oobFill);
}

// clang-format off
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/half.h>
#include <cutlass/numeric_types.h>
#include <cutlass/trace.h>
#include <cutlass/util/host_tensor.h>

// Rename the global function symbol
#define cuTensorMapEncodeTiled nvrtc_cuTensorMapEncodeTiled
#include <cute/tensor.hpp>
#undef cuTensorMapEncodeTiled
// Set everything back to normal

#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/epilogue/collective/collective_builder.hpp>

#include <cute/atom/mma_atom.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>
// clang-format on

#include <torch/csrc/distributed/c10d/cuda/cutlass/gemm/kernel/persistent_async_input_scheduler.cuh>

namespace {

using namespace cute;

template <typename LayoutB, typename TileShape_MNK, typename ClusterShape_MNK>
at::Tensor async_input_mm_impl(
    at::Tensor a,
    at::Tensor b,
    at::Tensor a_chunk_signals,
    int64_t a_chunk_pivot,
    at::Tensor out) {
  c10::cuda::CUDAGuard guard(a.device());

  using ElementA = cutlass::bfloat16_t;
  using LayoutA = cutlass::layout::RowMajor;
  constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;

  using ElementB = cutlass::bfloat16_t;
  constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;

  using ElementC = cutlass::bfloat16_t;
  using LayoutC = cutlass::layout::RowMajor;
  constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

  using ElementAccumulator = float;

  using KernelSchedule = cutlass::gemm::KernelTmaWarpSpecializedCooperative;
  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          TileShape_MNK,
          ClusterShape_MNK,
          cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAccumulator,
          ElementAccumulator,
          void,
          LayoutC,
          AlignmentC,
          ElementC,
          LayoutC,
          AlignmentC,
          EpilogueSchedule>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          ElementA,
          LayoutA,
          AlignmentA,
          ElementB,
          LayoutB,
          AlignmentB,
          ElementAccumulator,
          TileShape_MNK,
          ClusterShape_MNK,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          KernelSchedule>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue,
      cutlass::gemm::PersistentAsyncInputScheduler<KernelSchedule>>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;

  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && out.dim() == 2);
  TORCH_CHECK(a.is_contiguous() && out.is_contiguous());

  if constexpr (std::is_same_v<LayoutB, cutlass::layout::RowMajor>) {
    TORCH_CHECK(b.is_contiguous());
  } else {
    TORCH_CHECK(b.stride(1) == b.size(0));
    TORCH_CHECK(b.stride(0) == 1);
  }
  TORCH_CHECK_EQ(a.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(b.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(out.scalar_type(), at::kBFloat16);

  int M = static_cast<int>(a.sizes()[0]);
  int N = static_cast<int>(b.sizes()[1]);
  int K = static_cast<int>(a.sizes()[1]);
  TORCH_CHECK_EQ(b.sizes()[0], K);
  TORCH_CHECK_EQ(out.sizes()[0], M);
  TORCH_CHECK_EQ(out.sizes()[1], N);

  auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1});
  auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1});
  auto stride_C = cutlass::make_cute_packed_stride(StrideC{}, {M, N, 1});

  Gemm gemm;

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K, 1},
      {
          reinterpret_cast<ElementA*>(a.data_ptr<at::BFloat16>()),
          stride_A,
          reinterpret_cast<ElementB*>(b.data_ptr<at::BFloat16>()),
          stride_B,
      },
      {{1, 1},
       nullptr,
       stride_C,
       reinterpret_cast<ElementC*>(out.data_ptr<at::BFloat16>()),
       stride_C},
  };

  TORCH_CHECK(
      a_chunk_signals.sizes().size() == 1,
      "async_input_mm: `a_chunk_signals` must be a 1D tensor.");
  size_t num_chunks_M = a_chunk_signals.numel();

  TORCH_CHECK(
      M % num_chunks_M == 0,
      "async_input_mm: `a.shape(0)` must be an interger multiple of `a_chunk_signals.numel()`");
  size_t chunk_size_M = M / num_chunks_M;
  size_t tile_size_M = cute::get<0>(TileShape_MNK{});

  TORCH_CHECK(chunk_size_M % tile_size_M == 0);

  // We want to swizzle within a chunk
  arguments.scheduler.max_swizzle_size = chunk_size_M / tile_size_M;

  // PersistentAsyncInputScheduler currently only supports rastering along N
  using RasterOrderOptions = typename cutlass::gemm::kernel::detail::
      PersistentTileSchedulerSm90::RasterOrderOptions;
  arguments.scheduler.raster_order = RasterOrderOptions::AlongN;

  // Convert the number of chunks to pivot to the number of m idx to pivot
  arguments.scheduler.tile_idx_pivot_m =
      a_chunk_pivot * (chunk_size_M / tile_size_M);
  arguments.scheduler.tiles_per_chunk_m = chunk_size_M / tile_size_M;
  arguments.scheduler.chunk_signals = a_chunk_signals.data_ptr<uint32_t>();

  size_t workspace_size = Gemm::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  TORCH_CHECK(gemm.can_implement(arguments) == cutlass::Status::kSuccess);
  TORCH_CHECK(
      gemm.initialize(arguments, workspace.get()) == cutlass::Status::kSuccess);
  TORCH_CHECK(
      gemm(at::cuda::getCurrentCUDAStream()) == cutlass::Status::kSuccess);

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return out;
}

} // namespace

#endif

namespace c10d::cuda::detail {

#define DISPATCH_LAYOUT_B(is_b_row_major, ...)    \
  if (is_b_row_major) {                           \
    using LayoutB = cutlass::layout::RowMajor;    \
    __VA_ARGS__();                                \
  } else {                                        \
    using LayoutB = cutlass::layout::ColumnMajor; \
    __VA_ARGS__();                                \
  }

at::Tensor async_input_mm_out(
    at::Tensor a,
    at::Tensor b,
    at::Tensor a_chunk_signals,
    int64_t a_chunk_pivot,
    at::Tensor out) {
  TORCH_CHECK(
      a.dim() == 2 && b.dim() == 2 && out.dim() == 2,
      "async_input_mm: `a`, `b` and `out` must be matrices")
  TORCH_CHECK(
      a.is_contiguous() && out.is_contiguous(),
      "async_input_mm: `a` and `out` must be in row-major layout");

  bool is_b_row_major = b.is_contiguous();
  if (!b.is_contiguous()) {
    TORCH_CHECK(b.stride(1) == b.size(0));
    TORCH_CHECK(b.stride(0) == 1);
  }
  TORCH_CHECK_EQ(a.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(b.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(out.scalar_type(), at::kBFloat16);

  int64_t M = a.sizes()[0];
  int64_t N = b.sizes()[1];
  int64_t K = a.sizes()[1];
  TORCH_CHECK_EQ(b.sizes()[0], K);
  TORCH_CHECK_EQ(out.sizes()[0], M);
  TORCH_CHECK_EQ(out.sizes()[1], N);

#if defined(BUILD_ASYNC_MM_KERNEL)
  DISPATCH_LAYOUT_B(is_b_row_major, [&]() {
    // TODO(yifu): tuning
    async_input_mm_impl<LayoutB, Shape<_128, _256, _64>, Shape<_2, _1, _1>>(
        a, b, a_chunk_signals, a_chunk_pivot, out);
  });
#else
  TORCH_CHECK(
      false, "async_input_mm is not currenlty supported on your device");
#endif
  return out;
}

at::Tensor async_input_mm(
    at::Tensor a,
    at::Tensor b,
    at::Tensor a_chunk_signals,
    int64_t a_chunk_pivot) {
  TORCH_CHECK(
      a.dim() == 2 && b.dim() == 2,
      "async_input_mm: `a`, `b` and `out` must all be a matrix")

  int64_t M = a.sizes()[0];
  int64_t N = b.sizes()[1];
  auto out = a.new_empty({M, N});
  return async_input_mm_out(a, b, a_chunk_signals, a_chunk_pivot, out);
}

} // namespace c10d::cuda::detail
